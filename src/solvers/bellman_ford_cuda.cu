#include "hip/hip_runtime.h"

/*
 * This is a CUDA version of bellman_ford algorithm
 * Compile: nvcc -arch=sm_52 -o cuda_bellman_ford cuda_bellman_ford.cu
 * Run: ./cuda_bellman_ford <input file> <number of blocks per grid> <number of threads per block>, you will find the
 * output file 'output.txt'
 * */
#include <assert.h>
#include <stdbool.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "../generate_graphs/graph_generator.h"
#include "../generate_graphs/graph_structures.h"
#include "../generate_graphs/output_graphs.h"
#include "cuda_utils.h"
#include "output_structure.h"

#define INF 1000000

#include <stdio.h>
#ifdef _WIN32
#include <windows.h>
#else
#include <time.h>
#endif

double gettime(void) {
    /*#ifdef _WIN32
        LARGE_INTEGER frequency;
        LARGE_INTEGER start;
        QueryPerformanceFrequency(&frequency);
        QueryPerformanceCounter(&start);
        return (double)start.QuadPart / frequency.QuadPart;
    #else
    struct timespec ts;
    clock_gettime(CLOCK_MONOTONIC, &ts);
    return (ts.tv_sec + (double)ts.tv_nsec / 1e9);*/
    return 5.0;
    //#endif
}

__global__ void relax_initial(int *d_dist, int *d_predecessor, bool *d_wasUpdatedLastIter, bool *d_hasChanged, int n) {
    int bdim = blockDim.x, gdim = gridDim.x, bid = blockIdx.x, tid = threadIdx.x;
    int i = bdim * bid + tid;
    int skip = bdim * gdim;
    for (int k = i; k < n; k += skip) {
        d_dist[k] = INF;
        d_predecessor[k] = -1;
        d_wasUpdatedLastIter[k] = false;
        d_hasChanged[k] = false;
    }
    __syncthreads();
}

__global__ void copyHasChanged(bool *wasUpdatedLastIter, bool *hasChanged, int n) {
    int bdim = blockDim.x, gdim = gridDim.x, bid = blockIdx.x, tid = threadIdx.x;
    int i = bdim * bid + tid;
    int skip = bdim * gdim;

    for (int j = i; j < n; j += skip) {
        wasUpdatedLastIter[j] = hasChanged[j];
        wasUpdatedLastIter[j] = false;
    }
    __syncthreads();
}

__global__ void bellmanFordIteration(SourceEdge *outEdges, int outNeighbours, int *predecessor, int *dist,
                                     bool *wasUpdatedLastIter, bool *hasChanged, int source) {
    int bdim = blockDim.x, gdim = gridDim.x, bid = blockIdx.x, tid = threadIdx.x;
    int i = bdim * bid + tid;
    int skip = bdim * gdim;

    for (int edgeIndex = i; edgeIndex < outNeighbours; edgeIndex += skip) {
        if (*wasUpdatedLastIter) {
            int destination = outEdges[edgeIndex].dest;
            int weight = outEdges[edgeIndex].weight;
            int new_dist = dist[source] + weight;
            if (new_dist < dist[destination]) {
                hasChanged[destination] = true;
                dist[destination] = new_dist;
                predecessor[destination] = source;
            }
        }
    }
    __syncthreads();
}

/**
 * Bellman-Ford algorithm. Find the shortest path from vertex 0 to other vertices.
 * @param blockPerGrid number of blocks per grid
 * @param threadsPerBlock number of threads per block
 * @param n input size
 * @param *mat input adjacency matrix
 * @param *dist distance array
 * @param *has_negative_cycle a bool variable to recode if there are negative cycles
 */
BFOutput *bellmanFordCuda(int blocksPerGrid, int threadsPerBlock, SourceGraph *g, int startNode) {
    // Pointer to the graph on the device
    BFOutput *result;
    result = (BFOutput *)malloc(sizeof(BFOutput));
    (*result).startNode = startNode;
    (*result).predecessor = (int *)malloc((*g).numNodes * sizeof(int));
    (*result).dist = (int *)malloc((*g).numNodes * sizeof(int));
    (*result).negativeCycleNode = -1;
    (*result).numberNodes = (*g).numNodes;
    SourceGraph *d_graph;
    int *d_dist;
    int *d_predecessor;
    bool *d_wasUpdatedLastIter, *d_hasChanged;
    int n = (*g).numNodes;
    hipMalloc(&d_dist, n * sizeof(int));
    hipMalloc(&d_predecessor, n * sizeof(int));
    hipMalloc(&d_wasUpdatedLastIter, n * sizeof(bool));
    hipMalloc(&d_hasChanged, n * sizeof(bool));
    // Call the function to copy the SourceGraph to the GPU
    copySourceGraphToDevice(g, &d_graph);

    dim3 gdim(blocksPerGrid);
    dim3 bdim(threadsPerBlock);
    double tstart, tend;
    tstart = gettime();

    relax_initial<<<gdim, bdim>>>(d_dist, d_predecessor, d_wasUpdatedLastIter, d_hasChanged, n);

    hipDeviceSynchronize(); /* wait for kernel to finish */

    d_dist[startNode] = 0;
    d_wasUpdatedLastIter[startNode] = true;

    for (int iter = 0; iter < n; iter++) {
        for (int source = 0; source < n; ++source) {
            bellmanFordIteration<<<gdim, bdim>>>((*d_graph).nodes[source].outEdges,
                                                 (*d_graph).nodes[source].outNeighbours, d_predecessor, d_dist,
                                                 d_wasUpdatedLastIter, d_hasChanged, source);
            hipDeviceSynchronize();
            if (iter == n - 1 && d_hasChanged) {
                tend = gettime();
                hipMemcpy((*result).predecessor, d_predecessor, n * sizeof(int), hipMemcpyDeviceToHost);
                (*result).negativeCycleNode = source;
                (*result).hasNegativeCycle = true;
                (*result).timeInSeconds = tend - tstart;
                return result;
            }
        }
        copyHasChanged<<<gdim, bdim>>>(d_wasUpdatedLastIter, d_hasChanged, n);
        hipDeviceSynchronize();
    }

    tend = gettime();
    hipMemcpy((*result).dist, d_dist, n * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy((*result).predecessor, d_predecessor, n * sizeof(int), hipMemcpyDeviceToHost);
    (*result).hasNegativeCycle = false;
    (*result).timeInSeconds = tend - tstart;
    return result;
}

int main(int argc, char **argv) {
    int threadsPerBlock = 1024;
    SourceGraph *readGraph = readSourceGraphFromFile("../../data/no_cycle/graph_no_cycle_5.txt");
    int blocksPerGrid = ((*readGraph).numNodes + 1024) / 1024;

    BFOutput *result = bellmanFordCuda(blocksPerGrid, threadsPerBlock, readGraph, 0);
    printf("---------------- %d\n", (*result).hasNegativeCycle);
    writeResult(result, "../../results/omp_source/no_cycle/graph_no_cycle_5.txt", true);

    SourceGraph *readGraphNegativeCycle = readSourceGraphFromFile("../../data/cycle/graph_cycle_5.txt");
    BFOutput *resultCycle = bellmanFordCuda(blocksPerGrid, threadsPerBlock, readGraphNegativeCycle, 0);
    writeResult(resultCycle, "../../results/omp_source/cycle/graph_cycle_5.txt", true);
    return 0;
}